/*********************************************************************
* Filename:   rot-13.c
* Author:     Brad Conte (brad AT bradconte.com)
* Copyright:
* Disclaimer: This code is presented "as is" without any guarantees.
* Details:    Implementation of the ROT-13 encryption algorithm.
                  Algorithm specification can be found here:
                   *
                  This implementation uses little endian byte order.
*********************************************************************/

/*************************** HEADER FILES ***************************/
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>


int nWarps;


/*********************** FUNCTION DEFINITIONS ***********************/
__global__ void rot13Kernel(char str[], int numElements)
{
   int case_type;

   int idx = blockDim.x * blockIdx.x + threadIdx.x;

   if (idx < numElements) {
      // Only process alphabetic characters.
      if (!(str[idx] < 'A' || (str[idx] > 'Z' && str[idx] < 'a') || str[idx] > 'z')) {
         // Determine if the char is upper or lower case.
         if (str[idx] >= 'a')
            case_type = 'a';
         else
            case_type = 'A';
         // Rotate the char's value, ensuring it doesn't accidentally "fall off" the end.
         str[idx] = (str[idx] + 13) % (case_type + 26);
         if (str[idx] < 26)
            str[idx] += case_type;
      }
   }
}

void rot13(char answer[], char h_text[], int numElements) {
   // Error code to check return values for CUDA calls
   hipError_t err = hipSuccess;

   size_t size = numElements * sizeof(char);
   printf("string of %d elements\n", numElements);

   // Allocate the device input vector A
   char *d_text = NULL;
   err = hipMalloc((void **)&d_text, size);

   if (err != hipSuccess)
   {
     fprintf(stderr, "Failed to allocate device string (error code %s)!\n", hipGetErrorString(err));
     exit(EXIT_FAILURE);
   }

   // Copy the host input vectors A and B in host memory to the device input vectors in
   // device memory
   printf("Copy input data from the host memory to the CUDA device\n");
   err = hipMemcpy(d_text, h_text, size, hipMemcpyHostToDevice);

   if (err != hipSuccess)
   {
     fprintf(stderr, "Failed to copy string from host to device (error code %s)!\n", hipGetErrorString(err));
     exit(EXIT_FAILURE);
   }

   // Launch the Vector Add CUDA Kernel
   int threadsPerBlock = nWarps * 32;
   int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
   printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

   rot13Kernel<<<blocksPerGrid, threadsPerBlock>>>(d_text, numElements);
   err = hipGetLastError();

   if (err != hipSuccess)
   {
     fprintf(stderr, "Failed to launch rot13 kernel (error code %s)!\n", hipGetErrorString(err));
     exit(EXIT_FAILURE);
   }

   // Copy the device result vector in device memory to the host result vector
   // in host memory.
   printf("Copy output data from the CUDA device to the host memory\n");
   err = hipMemcpy(answer, d_text, size, hipMemcpyDeviceToHost);

   if (err != hipSuccess)
   {
     fprintf(stderr, "Failed to copy string from device to host (error code %s)!\n", hipGetErrorString(err));
     exit(EXIT_FAILURE);
   }

   // Free device global memory
   err = hipFree(d_text);

   if (err != hipSuccess)
   {
     fprintf(stderr, "Failed to free device string (error code %s)!\n", hipGetErrorString(err));
     exit(EXIT_FAILURE);
   }
}

int readTextFile(char filename[], char* str[]) {
   long length;
   FILE * f = fopen (filename, "rb");

   if (f)
   {
     fseek (f, 0, SEEK_END);
     length = ftell (f);

     fseek (f, 0, SEEK_SET);
     *str = (char*) malloc(length);
     if (str)
     {
       fread (*str, 1, length, f);
     }
     fclose (f);
   }

   return length;
}

// int main() {
//    char h_text[] = {"ABCDEFGHIJKLMNOPQRSTUVWXYZabcdefghijklmnopqrstuvwxyz"};
//    char h_code[] = {"NOPQRSTUVWXYZABCDEFGHIJKLMnopqrstuvwxyzabcdefghijklm"};



//    // Print the vector length to be used, and compute its size
//    int numElements = strlen(h_text);

//    size_t size = numElements * sizeof(char);

//    // Allocate the device answer string
//    char* answer = (char*) malloc(size);

//    rot13(answer, h_text, numElements);

//    if (strcmp(answer, h_code)) {
//       fprintf(stderr, "Result verification failed");
//       exit(EXIT_FAILURE);
//    }
//    printf("Test PASSED\n");

//    // Free host memory
//    free(answer);

//    printf("Done\n");
//    return 0;
// }

int main(int argc, char* argv[]) {
   char* h_text = NULL;
   char* filename = argv[1];
   sscanf(argv[2], "%d", &nWarps);

   printf("Reading file: %s\n", filename);

   // Print the vector length to be used, and compute its size
   int numElements = readTextFile(filename, &h_text);

   size_t size = numElements * sizeof(char);

   // Allocate the device answer string
   char* answer = (char*) malloc(size);
   char* original = (char*) malloc(size);

   rot13(answer, h_text, numElements);
   rot13(original, answer, numElements);

   if (strcmp(original, h_text)) {
      fprintf(stderr, "Result verification failed");
      exit(EXIT_FAILURE);
   }
   printf("Test PASSED\n");

   // Free host memory
   free(answer);

   printf("Done\n");
   return 0;
}